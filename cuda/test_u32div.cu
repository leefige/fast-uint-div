#include "hip/hip_runtime.h"
#include "u32div.cuh"

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <memory>
#include <random>
#include <thread>
#include <utility>
#include <vector>

#define CHECK_CUDA(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      printf("[CUDA Error] code %d at %s:%d: %s\n", err, __FILE__, __LINE__,   \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define CHECK_KERNEL()                                                         \
  do {                                                                         \
    hipError_t err = hipGetLastError();                                      \
    if (err != hipSuccess) {                                                  \
      printf("[CUDA Error] code %d at %s:%d: %s\n", err, __FILE__, __LINE__,   \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

static constexpr int HOST_THREAD_COUNT = 16;
static constexpr int CTA_SIZE = 256;
static constexpr int TEST_COUNT = 1 << 24;

template <int N, typename T> struct alignas(sizeof(uint4)) Vec {
  T data[N];
};

// one thread computes all elements
template <int BLOCK>
__global__ void __launch_bounds__(BLOCK, 1)
    kernel(uint32_t *out, const uint32_t *dividends, const U32Div div) {
  const uint32_t d = div.GetD();

  constexpr int UNROLL = 16 / sizeof(uint32_t);
  using VecT = Vec<UNROLL, uint32_t>;
  static_assert(sizeof(VecT) == sizeof(uint4),
                "assert sizeof(VecT) == sizeof(uint4)");
  static_assert(TEST_COUNT % (BLOCK * UNROLL) == 0,
                "requires TEST_COUNT % (BLOCK * UNROLL) == 0");

  VecT v_in;
  const VecT *in_ptr = reinterpret_cast<const VecT *>(dividends) +
                       blockIdx.x * BLOCK + threadIdx.x;
  v_in = *in_ptr;

  VecT v_out;
#pragma unroll
  for (int k = 0; k < UNROLL; ++k) {
    v_out.data[k] = v_in.data[k] / d;
  }

  VecT *out_ptr =
      reinterpret_cast<VecT *>(out) + blockIdx.x * BLOCK + threadIdx.x;
  *out_ptr = v_out;
  return;
}

template <typename Func>
void host_threading(std::vector<std::thread> &thds, Func &&func) {
  if (thds.size() < HOST_THREAD_COUNT) {
    thds.resize(HOST_THREAD_COUNT);
  }
  for (int i = 0; i < HOST_THREAD_COUNT; ++i) {
    thds[i] = std::thread(std::forward<Func>(func), i);
  }
  for (int i = 0; i < HOST_THREAD_COUNT; ++i) {
    thds[i].join();
  }
  return;
}

void test_body(const U32Div &div, bool large_n = false) {
  uint32_t d = div.GetD();

  float total_time_slow = 1;
  float total_time_fast = 1;

  std::vector<uint32_t> n_h(TEST_COUNT);
  std::vector<std::thread> thds(HOST_THREAD_COUNT);

  /* Step 1: generate dividends */
  static_assert(TEST_COUNT % HOST_THREAD_COUNT == 0,
                "requires TEST_COUNT % HOST_THREAD_COUNT == 0");
  constexpr int ELEM_PER_THREAD = TEST_COUNT / HOST_THREAD_COUNT;
  host_threading(thds, [&](int i) {
    using Dist = std::uniform_int_distribution<uint32_t>;
    std::default_random_engine rng(i);
    std::shared_ptr<Dist> dist;
    if (large_n) {
      dist = std::make_shared<Dist>(0, UINT32_MAX);
    } else {
      dist = std::make_shared<Dist>(0, INT32_MAX);
    }

    for (int j = 0; j < ELEM_PER_THREAD; ++j) {
      n_h[i * ELEM_PER_THREAD + j] = (*dist)(rng);
    }
  });

  /* Step 2: run host */
  std::vector<uint32_t> out_h(TEST_COUNT);
  host_threading(thds, [&](int i) {
    for (int j = 0; j < ELEM_PER_THREAD; ++j) {
      out_h[i * ELEM_PER_THREAD + j] = n_h[i * ELEM_PER_THREAD + j] / d;
    }
  });

  /* Step 3: copy dividends to device */
  uint32_t *n_d;
  CHECK_CUDA(hipMalloc(&n_d, n_h.size() * sizeof(uint32_t)));
  CHECK_CUDA(hipMemcpy(n_d, n_h.data(), n_h.size() * sizeof(uint32_t),
                        hipMemcpyHostToDevice));

  /* Step 4: run reference */
  std::vector<uint32_t> ref_h(TEST_COUNT);
  uint32_t *ref_d;
  CHECK_CUDA(hipMalloc(&ref_d, ref_h.size() * sizeof(uint32_t)));
  hipEvent_t ref_start, ref_stop;
  CHECK_CUDA(hipEventCreate(&ref_start));
  CHECK_CUDA(hipEventCreate(&ref_stop));

  static constexpr int CTA_COUNT =
      TEST_COUNT / (CTA_SIZE * 16 / sizeof(uint32_t));
  // warmup
  kernel<CTA_SIZE><<<CTA_COUNT, CTA_SIZE>>>(ref_d, n_d, div);
  CHECK_KERNEL();
  // run
  CHECK_CUDA(hipEventRecord(ref_start));
  kernel<CTA_SIZE><<<CTA_COUNT, CTA_SIZE>>>(ref_d, n_d, div);
  CHECK_KERNEL();
  CHECK_CUDA(hipEventRecord(ref_stop));
  CHECK_CUDA(hipEventSynchronize(ref_stop));
  CHECK_CUDA(hipEventElapsedTime(&total_time_slow, ref_start, ref_stop));
  CHECK_CUDA(hipEventDestroy(ref_start));
  CHECK_CUDA(hipEventDestroy(ref_stop));
  // check
  CHECK_CUDA(hipMemcpy(ref_h.data(), ref_d, ref_h.size() * sizeof(uint32_t),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(ref_d));
  host_threading(thds, [&](int i) {
    for (int j = 0; j < ELEM_PER_THREAD; ++j) {
      int idx = i * ELEM_PER_THREAD + j;
      if (out_h[idx] != ref_h[idx]) {
        printf("Error: %u / %u = %u, Ref returns: %u\n", n_h[idx], d,
               out_h[idx], ref_h[idx]);
        break;
      }
    }
  });

  // free
  CHECK_CUDA(hipFree(n_d));

  total_time_slow *= 1000;
  total_time_fast *= 1000;
  printf("d: %u,\tslow: %.2f us,\tfast: %.2f us,\tspeedup: %f\n", d,
         total_time_slow, total_time_fast, total_time_slow / total_time_fast);
  return;
}

int main() {
  srand((unsigned)time(nullptr));

  puts("DivBounded, d = rand() + 1");
  for (int i = 0; i < 5; i++) {
    uint32_t d = rand() + 1U;

    U32Div div(d);
    test_body(div);
  }

  return 0;
}
