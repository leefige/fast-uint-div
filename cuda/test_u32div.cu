#include "hip/hip_runtime.h"
#include "u32div.cuh"

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <memory>
#include <random>
#include <thread>
#include <utility>
#include <vector>

#define CHECK_CUDA(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      printf("[CUDA Error] code %d at %s:%d: %s\n", err, __FILE__, __LINE__,   \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

#define CHECK_KERNEL()                                                         \
  do {                                                                         \
    hipError_t err = hipGetLastError();                                      \
    if (err != hipSuccess) {                                                  \
      printf("[CUDA Error] code %d at %s:%d: %s\n", err, __FILE__, __LINE__,   \
             hipGetErrorString(err));                                         \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

static constexpr int HOST_THREAD_COUNT = 16;
static constexpr int CTA_SIZE = 256;
static constexpr int TEST_COUNT = 1 << 24;

namespace impl {

template <int N, typename T> struct alignas(sizeof(uint4)) Vec {
  T data[N];
};

struct DivideRef {
  __device__ __forceinline__ uint32_t operator()(uint32_t n,
                                                 const U32Div &div) const {
    return n / div.GetD();
  }
};

struct DivideBounded {
  __device__ __forceinline__ uint32_t operator()(uint32_t n,
                                                 const U32Div &div) const {
    return div.DivBounded(n);
  }
};

struct Divide {
  __device__ __forceinline__ uint32_t operator()(uint32_t n,
                                                 const U32Div &div) const {
    return div.Div(n);
  }
};

// one thread computes all elements
template <int BLOCK, typename Func>
__device__ __forceinline__ void kernel_impl(uint32_t *out,
                                            const uint32_t *dividends,
                                            const U32Div &div, Func &&func) {
  constexpr int UNROLL = 16 / sizeof(uint32_t);
  using VecT = Vec<UNROLL, uint32_t>;
  static_assert(sizeof(VecT) == sizeof(uint4),
                "assert sizeof(VecT) == sizeof(uint4)");
  static_assert(TEST_COUNT % (BLOCK * UNROLL) == 0,
                "requires TEST_COUNT % (BLOCK * UNROLL) == 0");

  VecT v_in;
  const VecT *in_ptr = reinterpret_cast<const VecT *>(dividends) +
                       blockIdx.x * BLOCK + threadIdx.x;
  v_in = *in_ptr;

  VecT v_out;
#pragma unroll
  for (int k = 0; k < UNROLL; ++k) {
    v_out.data[k] = func(v_in.data[k], div);
  }

  VecT *out_ptr =
      reinterpret_cast<VecT *>(out) + blockIdx.x * BLOCK + threadIdx.x;
  *out_ptr = v_out;
  return;
}

} // namespace impl

template <int BLOCK>
__global__ void __launch_bounds__(BLOCK, 1)
    kernel_reference(uint32_t *out, const uint32_t *dividends,
                     const U32Div div) {
  impl::kernel_impl<BLOCK>(out, dividends, div, impl::DivideRef());
}

template <int BLOCK>
__global__ void __launch_bounds__(BLOCK, 1)
    kernel_div(uint32_t *out, const uint32_t *dividends, const U32Div div) {
  impl::kernel_impl<BLOCK>(out, dividends, div, impl::Divide());
}

template <int BLOCK>
__global__ void __launch_bounds__(BLOCK, 1)
    kernel_div_bounded(uint32_t *out, const uint32_t *dividends,
                       const U32Div div) {
  impl::kernel_impl<BLOCK>(out, dividends, div, impl::DivideBounded());
}

class TestBase {
  static_assert(TEST_COUNT % HOST_THREAD_COUNT == 0,
                "requires TEST_COUNT % HOST_THREAD_COUNT == 0");

protected:
  virtual void launch_kernel(uint32_t *out, const uint32_t *dividends,
                             const U32Div &div) = 0;

  static constexpr int ELEM_PER_THREAD = TEST_COUNT / HOST_THREAD_COUNT;
  static constexpr int CTA_COUNT =
      TEST_COUNT / (CTA_SIZE * 16 / sizeof(uint32_t));

public:
  explicit TestBase(uint32_t d_, bool large_n_)
      : thds(HOST_THREAD_COUNT), n_h(TEST_COUNT), out_h(TEST_COUNT),
        ref_h(TEST_COUNT), target_h(TEST_COUNT), div(d_), total_time_slow(0),
        total_time_fast(0), large_n(large_n_) {
    setup();
  }
  virtual ~TestBase() { cleanup(); }

  void Run() {
    prelude();

    CHECK_CUDA(hipMemcpy(n_d, n_h.data(), n_h.size() * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    /* run reference */
    time_it(
        total_time_slow, ref_start, ref_stop, ref_h, ref_d, "reference", [&] {
          kernel_reference<CTA_SIZE><<<CTA_COUNT, CTA_SIZE>>>(ref_d, n_d, div);
        });

    /* run target */
    time_it(total_time_fast, target_start, target_stop, target_h, target_d,
            "target", [&] { launch_kernel(ref_d, n_d, div); });

    total_time_slow *= 1000;
    total_time_fast *= 1000;
    printf("d: %u,\treference: %.2f us,\ttarget: %.2f us,\tspeedup: %f\n",
           div.GetD(), total_time_slow, total_time_fast,
           total_time_slow / total_time_fast);
    return;
  }

private:
  template <typename Func> void host_threading(Func &&func) {
    if (thds.size() < HOST_THREAD_COUNT) {
      thds.resize(HOST_THREAD_COUNT);
    }
    for (int i = 0; i < HOST_THREAD_COUNT; ++i) {
      thds[i] = std::thread(std::forward<Func>(func), i);
    }
    for (int i = 0; i < HOST_THREAD_COUNT; ++i) {
      thds[i].join();
    }
    return;
  }

  void setup() {
    CHECK_CUDA(hipMalloc(&n_d, n_h.size() * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&ref_d, ref_h.size() * sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc(&target_d, target_h.size() * sizeof(uint32_t)));
    CHECK_CUDA(hipEventCreate(&ref_start));
    CHECK_CUDA(hipEventCreate(&ref_stop));
    CHECK_CUDA(hipEventCreate(&target_start));
    CHECK_CUDA(hipEventCreate(&target_stop));
  }

  void cleanup() {
    CHECK_CUDA(hipEventDestroy(target_start));
    CHECK_CUDA(hipEventDestroy(target_stop));
    CHECK_CUDA(hipEventDestroy(ref_start));
    CHECK_CUDA(hipEventDestroy(ref_stop));
    CHECK_CUDA(hipFree(target_d));
    CHECK_CUDA(hipFree(ref_d));
    CHECK_CUDA(hipFree(n_d));
  }

  void prelude() {
    /* generate dividends */
    host_threading([&](int i) {
      using Dist = std::uniform_int_distribution<uint32_t>;
      std::default_random_engine rng(i);
      std::shared_ptr<Dist> dist;
      if (large_n) {
        dist = std::make_shared<Dist>(0, UINT32_MAX);
      } else {
        dist = std::make_shared<Dist>(0, INT32_MAX);
      }

      for (int j = 0; j < ELEM_PER_THREAD; ++j) {
        n_h[i * ELEM_PER_THREAD + j] = (*dist)(rng);
      }
    });

    /* run host */
    host_threading([&](int i) {
      for (int j = 0; j < ELEM_PER_THREAD; ++j) {
        out_h[i * ELEM_PER_THREAD + j] =
            n_h[i * ELEM_PER_THREAD + j] / div.GetD();
      }
    });
  }

  template <typename Func>
  void time_it(float &duration, hipEvent_t start, hipEvent_t stop,
               std::vector<uint32_t> &data_host, uint32_t *data_device,
               const char *name, Func &&func) {
    // warmup
    func();
    CHECK_KERNEL();
    // run
    CHECK_CUDA(hipEventRecord(start));
    func();
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&duration, start, stop));
    CHECK_CUDA(hipMemcpy(data_host.data(), data_device,
                          data_host.size() * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    // check
    host_threading([&](int i) {
      for (int j = 0; j < ELEM_PER_THREAD; ++j) {
        int idx = i * ELEM_PER_THREAD + j;
        if (out_h[idx] != data_host[idx]) {
          printf("Error: %u / %u = %u, %s returns: %u\n", n_h[idx], div.GetD(),
                 out_h[idx], name, data_host[idx]);
          break;
        }
      }
    });
  }

  uint32_t *n_d;
  uint32_t *ref_d;
  uint32_t *target_d;

  hipEvent_t ref_start;
  hipEvent_t ref_stop;
  hipEvent_t target_start;
  hipEvent_t target_stop;

  std::vector<std::thread> thds;
  std::vector<uint32_t> n_h;
  std::vector<uint32_t> out_h;
  std::vector<uint32_t> ref_h;
  std::vector<uint32_t> target_h;

  U32Div div;
  float total_time_slow;
  float total_time_fast;
  bool large_n;
};

class TestDiv : public TestBase {
public:
  explicit TestDiv(uint32_t d_, bool large_n_ = false)
      : TestBase(d_, large_n_) {}

protected:
  virtual void launch_kernel(uint32_t *out, const uint32_t *dividends,
                             const U32Div &div) override {
    kernel_div<CTA_SIZE><<<CTA_COUNT, CTA_SIZE>>>(out, dividends, div);
  }
};

class TestDivBounded : public TestBase {
public:
  explicit TestDivBounded(uint32_t d_, bool large_n_ = false)
      : TestBase(d_, large_n_) {}

protected:
  virtual void launch_kernel(uint32_t *out, const uint32_t *dividends,
                             const U32Div &div) override {
    kernel_div_bounded<CTA_SIZE><<<CTA_COUNT, CTA_SIZE>>>(out, dividends, div);
  }
};

int main() {
  srand((unsigned)time(nullptr));

  puts("DivBounded, d = rand() + 1");
  for (int i = 0; i < 5; i++) {
    uint32_t d = rand() + 1U;

    TestDivBounded test(d, false);
    test.Run();
  }

  return 0;
}
